
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cublasXt.h>

int DEBUG = 1;

/* nvcc dot_product_cublasxt.cu -lcublas -o dot_product_cublasxt */

void initialize_rand(float *matrix, int A, int B){

   for(int i = 0; i < B; i++){   /* column */
      for(int j= 0 ; j < A; j++){   /* row */
         matrix[j * B + i] = (float)rand() / RAND_MAX;
      }
   }

}

void initialize_zero(float *matrix, int A, int B){

   for(int i = 0; i < B; i++){   /* column */
      for(int j = 0 ; j < A; j++){  /* row */
         matrix[j * B + i] = 0;
      }
   }

}

void matrix_print(float *matrix, int A, int B){

   for(int i = 0; i < B; i++){   /* column */
      for(int j= 0 ; j < A; j++){   /* row */
          printf("%3f ", matrix[j * B + i]);
      }
      printf("\n");
   }
   printf("\n");

}

int main(int argc, char *argv[]){

   clock_t start, end;
   hipEvent_t cu_start, cu_stop;
   float cuda_time = 0.0;

   hipEventCreate(&cu_start);
   hipEventCreate(&cu_stop);

   int N = 16, M = 16, L = 16;

   float alpha = 1.0, beta = 0.0;
   int lda, ldb, ldc;

   if(argc > 1){
      N = atoi(argv[1]);
   }

   if(argc > 2){
      M = atoi(argv[2]);
   }

   if(argc > 3){
      L = atoi(argv[3]);
   }

   if(argc > 4){
      DEBUG = atoi(argv[4]);
   }

   lda = N;
   ldb = M;
   ldc = N;

   float *host_matrixA;
   host_matrixA = (float *)malloc(sizeof(float) * N * M);

   float *host_matrixB;
   host_matrixB = (float *)malloc(sizeof(float) * M * L);

   float *host_matrixC;
   host_matrixC = (float *)malloc(sizeof(float) * N * L);

   srand(time(NULL));

   /* Initialize */
   initialize_rand(host_matrixA, N, M);
   initialize_rand(host_matrixB, M, L);
   initialize_zero(host_matrixC, N, L);

   /* Check */
   if(DEBUG >= 1){
      printf("matrixA\n");
      matrix_print(host_matrixA, N, M);
      printf("matrixB\n");
      matrix_print(host_matrixB, M, L);
   }

   start = clock();
   hipEventRecord(cu_start);

   cublasXtHandle_t handle;
   cublasXtCreate(&handle);

   /* Using GPU */
   int max_devices = 2;
   int devices[2] = {0, 1};
   cublasXtDeviceSelect(handle, max_devices, devices);

   /* Run */
   cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, L, M, &alpha, host_matrixA, lda, host_matrixB, ldb, &beta, host_matrixC, ldc);

   cublasXtDestroy(handle);
   hipEventRecord(cu_stop);
   hipEventSynchronize(cu_stop);
   hipEventElapsedTime(&cuda_time, cu_start, cu_stop);
   end = clock();

   /* Result */
   if(DEBUG >= 1){
      printf("matrixC = matrixA * matrixB\n");
      matrix_print(host_matrixC, N, L);
   }

   printf("Time: %f sec\n", (float)(end - start) / CLOCKS_PER_SEC);
   printf("Time(CUDA): %f sec\n", (cuda_time / 1000));

   free(host_matrixA);
   free(host_matrixB);
   free(host_matrixC);

}
